#include "hip/hip_runtime.h"
#include <stdexcept>
#include "array.cuh"


OrderedArray::OrderedArray(size_t capacity) {
    this->size = 0;
    this->capacity = capacity;
    hipMalloc(&data, sizeof(Particle) * capacity);
}

OrderedArray::~OrderedArray() {
    hipFree(data);
}

__global__ void get_particle_index_kernel(
                Particle *particles, size_t particle_id, uint *index)
{
    if (particles[threadIdx.x].id == particle_id)
        *index = threadIdx.x;
}

int OrderedArray::remove_by_id(size_t id) {
    uint *cudaIndex;
    hipMalloc(&cudaIndex, sizeof(uint));

    // TODO: Variable block size
    get_particle_index_kernel<<<1, size>>>(data, id, cudaIndex);

    uint *index = new uint;
    hipMemcpy(index, cudaIndex, sizeof(uint), hipMemcpyDeviceToHost);

    auto res = remove(*index);

    hipFree(cudaIndex);
    delete index;
    return res;
}

int OrderedArray::remove(size_t index) {
    if (index > size) {
        return INDEX_OUT_OF_RANGE;
    }

    if (index == size - 1) {
        --size;
        return 0;
    }

    auto parts_to_move = (size-(index+1));

    Particle *data_temp;
    hipMalloc(&data_temp, sizeof(Particle)*parts_to_move);

    hipMemcpy(data_temp, &data[index+1], parts_to_move*sizeof(Particle), hipMemcpyDeviceToDevice);
    hipMemcpy(&data[index], data_temp, parts_to_move*sizeof(Particle), hipMemcpyDeviceToDevice);

    --size;

    hipFree(data_temp);
    return 0;
}

// this is helper function for debugging, it prints all elements in particles array
__global__ void print_kernel(Particle *particles, size_t size) {
    for (int i = 0; i < size; i++) {
        printf("particle[%i]: %f %f %f %f\n", i, particles[i].x, particles[i].y,
                                particles[i].z, particles[i].sigma);
    }
}

int OrderedArray::insert(Particle value, size_t index) {
    if (index > size) {
        return INDEX_OUT_OF_RANGE;
    }

    auto parts_to_move = (size-index);

    Particle *data_temp;
    hipMalloc(&data_temp, sizeof(Particle)*parts_to_move);

    hipMemcpy(data_temp, &data[index], parts_to_move*sizeof(Particle), hipMemcpyDeviceToDevice);
    hipMemcpy(&data[index+1], data_temp, parts_to_move*sizeof(Particle), hipMemcpyDeviceToDevice);

    hipMemcpy(&data[index], &value, sizeof(Particle), hipMemcpyHostToDevice);

    ++size;

    hipFree(data_temp);
    return 0;
}

const Particle *OrderedArray::get_array() {
    return data;
}

size_t OrderedArray::getSize() const {
    return size;
}

void OrderedArray::set_data(Particle *data, size_t size) {
    if (size > capacity) {
        throw std::runtime_error("Something went wrong when setting OrderedArray on GPU:\
                size of Particle array is greater than capacity.");
    }
    hipFree(this->data);
    hipMalloc(&this->data, sizeof(Particle) * capacity);
    hipMemcpy(this->data, data, sizeof(Particle) * size, hipMemcpyHostToDevice);

    this->size = size;
}
