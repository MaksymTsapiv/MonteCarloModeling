#include "hip/hip_runtime.h"
#include <stdexcept>
#include "array.cuh"


OrderedArray::OrderedArray(size_t capacity) {
    this->size = 0;
    this->capacity = capacity;
    hipMalloc(&data, sizeof(Particle) * capacity);
}

OrderedArray::~OrderedArray() {
    hipFree(data);
}

int OrderedArray::remove(size_t index) {
    if (index > size) {
        return INDEX_OUT_OF_RANGE;
    }
    if (index == size - 1) {
        --size;
        return 0;
    }
    for (size_t i = index; i < size - 1; ++i) {
        data[i] = data[i + 1];
    }
    --size;

    return 0;
}

// this is helper function for debugging, it prints all elements in particles array
__global__ void print_kernel(Particle *particles, size_t size) {
    for (int i = 0; i < size; i++) {
        printf("particle[%i]: %f %f %f %f\n", i, particles[i].x, particles[i].y,
                                particles[i].z, particles[i].sigma);
    }
}

int OrderedArray::insert(Particle value, size_t index) {
    if (index > size) {
        return INDEX_OUT_OF_RANGE;
    }
    for (size_t i = index; i < size; ++i) {
        data[i] = data[i + 1];
    }
    hipMemcpy(&data[index], &value, sizeof(Particle), hipMemcpyHostToDevice);
    ++size;
    return 0;
}

const Particle *OrderedArray::get_array() {
    return data;
}

size_t OrderedArray::getSize() {
    return size;
}
