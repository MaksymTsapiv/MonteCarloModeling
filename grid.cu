#include "hip/hip_runtime.h"
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++, C#, and Java: http://www.viva64.com

#include <fstream>
#include <sstream>
#include <stdexcept>
#include <string>
#include <cstddef>
#include <random>
#include <cmath>
#include <vector>
#include <iostream>
#include "cell.cuh"
#include "grid.cuh"
#include "particle.cuh"

__host__ __device__ double Grid::get_Lx() const{
    return Lx;
}
__host__ __device__ double Grid::get_Ly() const{
    return Ly;
}
__host__ __device__ double Grid::get_Lz() const{
    return Lz;
}

void Grid::set_Lx(double x) {
    Lx = x;
}
void Grid::set_Ly(double y) {
    Ly = y;
}
void Grid::set_Lz(double z) {
    Lz = z;
}

void Grid::common_initializer(double x, double y, double z){
    cells.reserve(dim_cells.x * dim_cells.y * dim_cells.z);

    for (int i = 0; i < dim_cells.x; i++) {
        for (int j = 0; j < dim_cells.y; j++) {
            for (int k = 0; k < dim_cells.z; k++) {
                cells.emplace_back(i, j, k);
            }
        }
    }

    Lx = x;
    Ly = y;
    Lz = z;
    adj_cells = compute_adj_cells();
}

double random_double(double from, double to) {
    std::random_device rd;
    std::mt19937 rand_double(rd());

    std::uniform_real_distribution<> dist(from, to);
    return dist(rand_double);
}


__host__ __device__ double calc_dist(Particle p1, Particle p2) {
    double x1 = p1.get_x();
    double x2 = p2.get_x();
    double y1 = p1.get_y();
    double y2 = p2.get_y();
    double z1 = p1.get_z();
    double z2 = p2.get_z();

    return hypot(hypot(x1 - x2, y1 - y2), z1 - z2);
    // return sqrt(pow(sqrt(pow((x1 - x2), 2) + pow((y1 - y2), 2)), 2), pow((z1 -z2), 2));
}

size_t Grid::get_cell_id(double x, double y, double z) const {
    auto x_cell = static_cast<size_t>(floor(x / Lx * dim_cells.x));
    auto y_cell = static_cast<size_t>(floor(y / Ly * dim_cells.y));
    auto z_cell = static_cast<size_t>(floor(z / Lz * dim_cells.z));

    return x_cell + y_cell * dim_cells.y + z_cell * dim_cells.z * dim_cells.z;
};

__host__ __device__ Particle Grid::get_particle(size_t id) {
    for (auto particle : particles) {
        if (particle.get_id() == id) {
            return particle;
        }
    }
    return {};
}

void Grid::fill(size_t n) {
    bool flag = true;
    size_t count_tries = 0;
    size_t max_tries = 10000 + n;

    double sigma = 1.0;

    while ((particles.size() < n) && count_tries < max_tries) {

        double x = Lx * random_double(0, 1);
        double y = Ly * random_double(0, 1);
        double z = Lz * random_double(0, 1);

        Particle particle = Particle(x, y, z, sigma);

        for (auto &cell : cells) {
            for (auto pid : cell.get_particles()) {
                if (calc_dist(get_particle(pid), particle) <= sigma) {
                    flag = false;
                    break;
                }
            }
        }

        if (flag) {
            particles.push_back(particle);
            cells[get_cell_id(x, y, z)].add_particle(particle.get_id());
        }
        flag = true;

        count_tries++;
    }
}

// This is temporary function just to make it work. TODO: make up a better design
__device__ double calc_dist(double x1, double y1, double z1, double x2, double y2, double z2) {
    return sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2) + pow(z1 - z2, 2));
}

__global__ void parent_kernel(Particle *p1, Particle *p, D3 grid_size, bool *intersects) {
    if (p1->get_id() == p[threadIdx.x].get_id()) {
        intersects[threadIdx.x] = false;
        return;
    }

    auto sigma = p1->get_sigma();

    auto x1 = p1->get_x();
    auto y1 = p1->get_y();
    auto z1 = p1->get_z();

    auto x2 = p[threadIdx.x].get_x();
    auto y2 = p[threadIdx.x].get_y();
    auto z2 = p[threadIdx.x].get_z();

    if (x1 >= grid_size.x/2)
        x1 -= grid_size.x;
    if (x2 >= grid_size.x/2)
        x2 -= grid_size.x;

    if (y1 >= grid_size.y/2)
        y1 -= grid_size.y;
    if (y2 >= grid_size.y/2)
        y2 -= grid_size.y;

    if (z1 >= grid_size.z/2)
        z1 -= grid_size.z;
    if (z2 >= grid_size.z/2)
        z2 -= grid_size.z;

    if (calc_dist(x1, y1, z1, x2, y2, z2) <= sigma) {
        intersects[threadIdx.x] = true;
        return;
    }
    intersects[threadIdx.x] = false;
}

void Grid::move(double dispmax) {
    size_t count = 0;
    double sigma = 1.0;

    for (auto & particle : particles) {
        double new_x = particle.get_x() + random_double(-1, 1);
        double new_y = particle.get_y() + random_double(-1, 1);
        double new_z = particle.get_z() + random_double(-1, 1);

        double vec_x = new_x - particle.get_x();
        double vec_y = new_y - particle.get_y();
        double vec_z = new_z - particle.get_z();

        double vec_length = sqrt(pow(vec_x, 2) + pow(vec_y, 2) + pow(vec_z, 2));

        vec_x = vec_x / vec_length;
        vec_y = vec_y / vec_length;
        vec_z = vec_z / vec_length;

        double x = particle.get_x() + vec_x * dispmax;
        double y = particle.get_y() + vec_y * dispmax;
        double z = particle.get_z() + vec_z * dispmax;

        if (x >= Lx) x -= Lx;
        if (y >= Ly) y -= Ly;
        if (z >= Lz) z -= Lz;

        if (x < 0) x += Lx;
        if (y < 0) y += Ly;
        if (z < 0) z += Lz;

        std::vector<Particle> all_particles;
        for (auto cell_id : adj_cells[get_cell_id(particle.get_x(), particle.get_y(), particle.get_z())]) {
            const auto &cell = cells[cell_id];
            for (auto pid : cell.get_particles()) {
                if (get_particle(pid).get_id() == particle.get_id())
                    continue;

                all_particles.push_back(get_particle(pid));
            }
        }

        Particle particle_new = Particle(x, y, z, sigma);
        Particle *particle_cuda;
        hipMalloc(&particle_cuda, sizeof(Particle));
        hipMemcpy(particle_cuda, &particle_new, sizeof(Particle), hipMemcpyHostToDevice);

        auto p_size = all_particles.size();

        Particle *all_particles_cuda;
        hipMalloc(&all_particles_cuda, sizeof(Particle) * p_size);
        hipMemcpy(all_particles_cuda, all_particles.data(), sizeof(Particle) * p_size, hipMemcpyHostToDevice);

        bool *intersect_status;
        hipMalloc(&intersect_status, sizeof(bool)*p_size);

        parent_kernel<<<1, p_size>>>(particle_cuda, all_particles_cuda, dim_cells, intersect_status);

        bool *intersect_status_host = (bool *) malloc(sizeof(bool) * p_size);
        hipMemcpy(intersect_status_host, intersect_status, sizeof(bool)*p_size, hipMemcpyDeviceToHost);

        bool not_intersected = true;
        for (size_t i = 0; i < p_size; i++) {
            if (intersect_status_host[i]) {
                not_intersected = false;
                break;
            }
        }

        if (not_intersected) {
            particle.set_x(x);
            particle.set_y(y);
            particle.set_z(z);
        }
    }
}

enum paramsMLen{
    TYPE_MLEN = 6, SN_MLEN = 5, NAME_MLEN = 4, ALT_LOC_IND_MLEN = 1, RES_NAME_MLEN = 3,
    CHAIN_IND_MLEN = 1, RES_SEQ_NUM_MLEN = 4, RES_INS_CODE_MLEN = 1,
    X_MLEN = 8, Y_MLEN = 8, Z_MLEN = 8, OCC_MLEN = 6, TEMP_FACTOR_MLEN = 6,
    SEG_ID_MLEN = 4, ELEM_SYMB_MLEN = 2, CHARGE_MLEN = 2
};

static std::string
format(double fp_num, unsigned nint, unsigned nfrac) {
    auto maxNum = std::pow(10, nint);
    if (fp_num >= maxNum)
        throw std::invalid_argument(std::string("Number is too big (max ")
                                    + std::to_string(maxNum) + std::string(")"));

    fp_num = std::ceil(fp_num * maxNum) / static_cast<double>(maxNum);

    std::stringstream fp_num_ss;
    fp_num_ss.precision(nfrac);
    fp_num_ss.setf(std::ios::fixed, std::ios::floatfield);
    fp_num_ss << fp_num;

    return fp_num_ss.str();
}

constexpr auto COORD_MINT = 4;
constexpr auto COORD_MFRAC = 3;

constexpr auto OCCTEMP_MINT = 3;
constexpr auto OCCTEMP_MFRAC = 2;

static std::string
fcoord (double coord) {
    return format(coord, COORD_MINT, COORD_MFRAC);
}

static std::string
focctemp (double occtemp) {
    return format(occtemp, OCCTEMP_MINT, OCCTEMP_MFRAC);
}

enum direction{left, right};

static std::string check_fill (std::string val, size_t len, direction align) {
    auto val_len = val.size();
    if (val_len == 0)
        for (auto i = len; i > 0; i--, val += " ");
    else if (val_len > len)
        throw std::invalid_argument("Invalid argument length (too long): expected " +
                                    std::to_string(len) + ", got " + std::to_string(val_len));
    else {
        std::string xfix;
        for (auto i = val.size(); i < len; i++, xfix += " ");
        val = (align == right) ? xfix + val : val + xfix;
    }
    return val;
}

static std::string
check_fill(std::string val, int len) {
    return check_fill(val, len, left);
}

static void
export_to_pdb ( std::string fn,             // output filename with extension
                std::string type,           // 1-6
                std::string sn,             // 7-11  right
                std::string name,           // 13-16
                std::string alt_loc_ind,    // 17
                std::string res_name,       // 18-20 right
                std::string chain_ind,      // 22
                std::string res_seq_num,    // 23-26 right
                std::string res_ins_code,   // 27
                std::string x,              // 31-38 right
                std::string y,              // 39-46 right
                std::string z,              // 47-54 right
                std::string occ,            // 55-60 right
                std::string temp_factor,    // 61-66 right
                std::string seg_id,         // 73-76
                std::string elem_symb,      // 77-78 right
                std::string charge          // 79-80
              ){

    type = check_fill(type, TYPE_MLEN);
    sn = check_fill(sn, SN_MLEN, right);
    name = check_fill(name, NAME_MLEN);
    alt_loc_ind = check_fill(alt_loc_ind, ALT_LOC_IND_MLEN);
    res_name = check_fill(res_name, RES_NAME_MLEN, right);
    chain_ind = check_fill(chain_ind, CHAIN_IND_MLEN);
    res_seq_num = check_fill(res_seq_num, RES_SEQ_NUM_MLEN, right);
    res_ins_code = check_fill(res_ins_code, RES_INS_CODE_MLEN);
    x = check_fill(x, X_MLEN, right);
    y = check_fill(y, Y_MLEN, right);
    z = check_fill(z, Z_MLEN, right);
    occ = check_fill(occ, OCC_MLEN, right);
    temp_factor = check_fill(temp_factor, TEMP_FACTOR_MLEN, right);
    seg_id = check_fill(seg_id, SEG_ID_MLEN);
    elem_symb = check_fill(elem_symb, ELEM_SYMB_MLEN, right);
    charge = check_fill(charge, CHARGE_MLEN);

    std::ofstream pdb_file(fn, std::ofstream::app);
    pdb_file << type << sn << " " << name << alt_loc_ind << res_name << " " << chain_ind
             << res_seq_num << res_ins_code << "   " << x << y << z << occ << temp_factor
             << "     " << elem_symb << charge << std::endl;
    pdb_file.close();
}

void Grid::export_to_pdb(std::string fn) {
    remove(fn.data());
    unsigned serial_num = 1;
    for (auto particle : particles) {

        std::string sn_str = std::to_string(serial_num);

        const std::string particle_type = "ATOM";
        const std::string atom_name = "C";
        const std::string sort_of_elem = std::to_string(1);
        const std::string temp_factor = focctemp(0);

        ::export_to_pdb(fn, particle_type, std::to_string(serial_num), atom_name, "", "", "", sort_of_elem, "",
                fcoord(particle.get_x()), fcoord(particle.get_y()), fcoord(particle.get_z()),
                focctemp(particle.get_sigma()), temp_factor, "", "", "");
        serial_num++;
    }
}

/*
 * Find and return map where keys are cells and values are adjacent cells (excluding the key cell)
 */
std::map<size_t, std::vector<size_t>> Grid::compute_adj_cells() const {

    std::map<size_t, std::vector<size_t>> adj_cells;

    for (auto i = 0; i < dim_cells.x; i++)
    {
        int li = i == 0 ? dim_cells.x - 1 : i - 1;
        int ri = i == dim_cells.x - 1 ? 0 : i+1;
        for (auto j = 0; j < dim_cells.y; j++)
        {
            int lj = j == 0 ? dim_cells.y - 1 : j - 1;
            int rj = j == dim_cells.y - 1 ? 0 : j+1;
            for (auto k = 0; k < dim_cells.z; k++)
            {
                int lk = k == 0 ? dim_cells.z - 1 : k - 1;
                int rk = k == dim_cells.z - 1 ? 0 : k+1;

                adj_cells[get_cell_id(i, j, k)].push_back(get_cell_id(i, j, k));    // self

                adj_cells[get_cell_id(li, j, k)].push_back(get_cell_id(i, j, k));   // left on x axis
                adj_cells[get_cell_id(i, lj, k)].push_back(get_cell_id(i, j, k));   // left on y axis
                adj_cells[get_cell_id(i, j, lk)].push_back(get_cell_id(i, j, k));   // left on z axis

                adj_cells[get_cell_id(ri, j, k)].push_back(get_cell_id(i, j, k));   // right on x axis
                adj_cells[get_cell_id(i, rj, k)].push_back(get_cell_id(i, j, k));   // right on y axis
                adj_cells[get_cell_id(i, j, rk)].push_back(get_cell_id(i, j, k));   // right on z axis

                adj_cells[get_cell_id(li, lj, k)].push_back(get_cell_id(i, j, k));
                adj_cells[get_cell_id(li, j, lk)].push_back(get_cell_id(i, j, k));
                adj_cells[get_cell_id(i, lj, lk)].push_back(get_cell_id(i, j, k));

                adj_cells[get_cell_id(li, rj, k)].push_back(get_cell_id(i, j, k));
                adj_cells[get_cell_id(li, j, rk)].push_back(get_cell_id(i, j, k));
                adj_cells[get_cell_id(ri, lj, k)].push_back(get_cell_id(i, j, k));
                adj_cells[get_cell_id(ri, j, rk)].push_back(get_cell_id(i, j, k));
                adj_cells[get_cell_id(i, rj, lk)].push_back(get_cell_id(i, j, k));
                adj_cells[get_cell_id(i, lj, rk)].push_back(get_cell_id(i, j, k));

                adj_cells[get_cell_id(ri, rj, k)].push_back(get_cell_id(i, j, k));
                adj_cells[get_cell_id(ri, j, rk)].push_back(get_cell_id(i, j, k));
                adj_cells[get_cell_id(i, rj, rk)].push_back(get_cell_id(i, j, k));

                adj_cells[get_cell_id(li, lj, lk)].push_back(get_cell_id(i, j, k));
                adj_cells[get_cell_id(ri, rj, rk)].push_back(get_cell_id(i, j, k));

                adj_cells[get_cell_id(li, lj, rk)].push_back(get_cell_id(i, j, k));
                adj_cells[get_cell_id(li, rj, lk)].push_back(get_cell_id(i, j, k));
                adj_cells[get_cell_id(ri, lj, lk)].push_back(get_cell_id(i, j, k));

                adj_cells[get_cell_id(ri, rj, lk)].push_back(get_cell_id(i, j, k));
                adj_cells[get_cell_id(ri, lj, rk)].push_back(get_cell_id(i, j, k));
                adj_cells[get_cell_id(li, rj, rk)].push_back(get_cell_id(i, j, k));
            }
        }
    }

    // 100 * 100 * 100 * 27 * 4

    // print ajd_cells map
//    for (auto & adj_cell : adj_cells) {
//        std::cout << adj_cell.first << ": ";
//        for (int & it2 : adj_cell.second) {
//            std::cout << it2 << " ";
//        }
//        std::cout << std::endl;
//    }

    return adj_cells;
}
