#include "hip/hip_runtime.h"
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++, C#, and Java: http://www.viva64.com

#include <fstream>
#include <sstream>
#include <stdexcept>
#include <string>
#include <cstddef>
#include <random>
#include <cmath>
#include <vector>
#include <iostream>
#include <algorithm>

#include "grid.cuh"
#include "particle.cuh"
#include "time_measurement.cuh"

__host__ __device__ double Grid::get_Lx() const{
    return L.x;
}
__host__ __device__ double Grid::get_Ly() const{
    return L.y;
}
__host__ __device__ double Grid::get_Lz() const{
    return L.z;
}

__host__ __device__ D3<double> Grid::get_L() const {
    return L;
}

void Grid::set_Lx(double x) {
    L.x = x;
}
void Grid::set_Ly(double y) {
    L.y = y;
}
void Grid::set_Lz(double z) {
    L.z = z;
}

double random_double(double from, double to) {
    //std::random_device rd;
    //static std::mt19937 rand_double(rd());

    static std::mt19937 rand_double(1);

    std::uniform_real_distribution<> dist(from, to);
    return dist(rand_double);
}


__host__ __device__ double calc_dist(Particle p1, Particle p2) {
    double x1 = p1.x;
    double x2 = p2.x;
    double y1 = p1.y;
    double y2 = p2.y;
    double z1 = p1.z;
    double z2 = p2.z;

    return hypot(hypot(x1 - x2, y1 - y2), z1 - z2);
    // return sqrt(pow(sqrt(pow((x1 - x2), 2) + pow((y1 - y2), 2)), 2), pow((z1 -z2), 2));
}

std::vector<Particle> Grid::get_particles() const {
    return particles;
}

Particle Grid::get_particle(uint id) const {
    for (auto p : particles) {
        if (p.id == id) {
            return p;
        }
    }
    return {};
}

double Grid::volume() const {
    return L.x * L.y * L.z;
}

size_t Grid::n_particles() const {
    return particles.size();
}

double Grid::density() const {
    return n_particles() / volume();
}

double Grid::distance(int id1, int id2) const {
    auto x_dist = std::min(fabs(get_particle(id1).x - get_particle(id2).x),
            L.x - fabs(get_particle(id1).x - get_particle(id2).x));

    auto y_dist = std::min(fabs(get_particle(id1).y - get_particle(id2).y),
            L.y - fabs(get_particle(id1).y - get_particle(id2).y));

    auto z_dist = std::min(fabs(get_particle(id1).z - get_particle(id2).z),
            L.z - fabs(get_particle(id1).z - get_particle(id2).z));

    return sqrt(x_dist*x_dist + y_dist*y_dist + z_dist*z_dist);
}

__device__ double device_min(double a, double b) {
    return a < b ? a : b;
}

__global__ void
check_intersect (
        const Particle *particle,
        const Particle *ordered_particles,
        const uint *cellStartIdx,
        uint curr_cell_id,
        const D3<double> *L,
        int *intersects) {

    uint startIdx = cellStartIdx[curr_cell_id];
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    auto xd = device_min( fabs(particle->x - ordered_particles[startIdx+idx].x),
                        L->x - fabs(particle->x - ordered_particles[startIdx+idx].x) );

    auto yd = device_min( fabs(particle->y - ordered_particles[startIdx+idx].y),
                        L->y - fabs(particle->y - ordered_particles[startIdx+idx].y) );

    auto zd = device_min( fabs(particle->z - ordered_particles[startIdx+idx].z),
                        L->z - fabs(particle->z - ordered_particles[startIdx+idx].z) );

    auto dist = hypot(hypot(xd, yd), zd);
    if (dist < particle->sigma)
        atomicAdd(intersects, 1);
}

__global__ void update_kernel(uint *cellStartIdx, size_t cell_idx) {
    cellStartIdx[cell_idx+threadIdx.x]++;
}

void Grid::fill() {
    size_t count_tries = 0;
    size_t max_tries = 10000 * n;

    double sigma = 1.0;

    long long fors_time = 0;
    long long add_time = 0;

    while ((particles.size() < n) && count_tries < max_tries) {

        double x = L.x * random_double(0, 1);
        double y = L.y * random_double(0, 1);
        double z = L.z * random_double(0, 1);

        Particle particle = Particle(x, y, z, sigma);

        Particle *cuda_particle;
        hipMalloc(&cuda_particle, sizeof(Particle));
        hipMemcpy(cuda_particle, &particle, sizeof(Particle), hipMemcpyHostToDevice);

        D3<double> p_point = particle.get_coord();
        D3<int> p_cell = get_cell(p_point);

        bool intersected = false;

        auto for_start = get_current_time_fenced();
        for (auto z_off = -1; z_off <= 1; ++z_off) {
            for (auto y_off = -1; y_off <= 1; ++y_off) {
                for (auto x_off = -1; x_off <= 1; ++x_off) {
                    D3<int> offset = {x_off, y_off, z_off};
                    uint curr_cell_id = cell_id(p_cell + offset);

                    // number of particles in cell
                    size_t partInCell = partPerCell[curr_cell_id];

                    if (partInCell == 0)
                        continue;

                    const Particle *cuda_ordered_particles = particles_ordered.get_array();
                    // TODO: Variable block size
                    check_intersect<<<1, partInCell>>>( cuda_particle, cuda_ordered_particles,
                                                cellStartIdx, curr_cell_id, cudaL, intersectsCuda );

                    int *intersects = new int;
                    hipMemcpy(intersects, intersectsCuda, sizeof(int),
                                                            hipMemcpyDeviceToHost);

                    if (*intersects > 0)
                        intersected = true;

                    hipMemset(intersectsCuda, 0, sizeof(int));

                    delete intersects;
                }
                if (intersected) break;
            }
            if (intersected) break;
        }
        auto for_end = get_current_time_fenced();
        fors_time += to_us(for_end - for_start);

        auto add_start = get_current_time_fenced();
        if (!intersected) {
            particles.push_back(particle);
            if (particles.size() % 1000 == 0) std::cout << "size = " << particles.size() << '\n';
            auto cell_idx = cell_id(p_cell);

            // Cell start index in ordered array for the current particle (which is inserted)
            uint *partCellStartIdx = new uint;
            hipMemcpy(partCellStartIdx, &cellStartIdx[cell_idx], sizeof(uint),
                                                        hipMemcpyDeviceToHost);

            auto add_start2 = get_current_time_fenced();

            particles_ordered.insert(particle, *partCellStartIdx);
            partPerCell[cell_idx]++;

            auto add_end2 = get_current_time_fenced();

            // TODO: Variable block size
            if (static_cast<int>(n_cells-cell_idx-1) > 0)
                update_kernel<<<1, n_cells-cell_idx-1>>>(cellStartIdx, cell_idx+1);
        }
        auto add_end = get_current_time_fenced();
        add_time += to_us(add_end - add_start);

        count_tries++;
        hipFree(cuda_particle);
    }
    std::cout << "Tries: " << count_tries << std::endl;

    std::cout << "Fors time: " << fors_time << std::endl;
    std::cout << "Add time:  " << add_time << std::endl << std::endl;

    std::cout << std::endl;
}

void Grid::move(double dispmax) {
//    size_t count = 0;
//    double sigma = 1.0;

    for (auto & particle : particles) {
        double new_x = particle.x + random_double(-1, 1);
        double new_y = particle.y + random_double(-1, 1);
        double new_z = particle.z + random_double(-1, 1);

        double vec_x = new_x - particle.x;
        double vec_y = new_y - particle.y;
        double vec_z = new_z - particle.z;

        double vec_length = sqrt(pow(vec_x, 2) + pow(vec_y, 2) + pow(vec_z, 2));

        vec_x = vec_x / vec_length;
        vec_y = vec_y / vec_length;
        vec_z = vec_z / vec_length;

        double x = particle.x + vec_x * dispmax;
        double y = particle.y + vec_y * dispmax;
        double z = particle.z + vec_z * dispmax;

        if (x >= L.x) x -= L.x;
        if (y >= L.y) y -= L.y;
        if (z >= L.z) z -= L.z;

        if (x < 0) x += L.x;
        if (y < 0) y += L.y;
        if (z < 0) z += L.z;

        // TODO: implement
        // PROBLEM: with this approach we will have to iterate through the array of bools to
        //    check if any thread of kernel function returned true
        // Calculate <new_particle_coord_cell_id> -- cell id of the new particle position
        for (int z = -1; z <= 1; z++)
            for (int y = -1; y <= 1; y++)
                for (int x = -1; x <= 1; x++) {
                    // Get current cell id <curr_cell_id>, relative to <new_particle_coord_cell_id>
                    // Parallel check for intersect in <curr_cell_id>, passing <ordered_array>,
                    //    start index and end index for the <ordered_array> for current cell
                    
                    // Kernel function saves result in array of bools <intersects>
                    // Check the array <intersects>
                }

        bool not_intersected = true;
        if (not_intersected) {
            particle.x = x;
            particle.y = y;
            particle.z = z;
        }
    }
}

enum paramsMLen{
    TYPE_MLEN = 6, SN_MLEN = 5, NAME_MLEN = 4, ALT_LOC_IND_MLEN = 1, RES_NAME_MLEN = 3,
    CHAIN_IND_MLEN = 1, RES_SEQ_NUM_MLEN = 4, RES_INS_CODE_MLEN = 1,
    X_MLEN = 8, Y_MLEN = 8, Z_MLEN = 8, OCC_MLEN = 6, TEMP_FACTOR_MLEN = 6,
    SEG_ID_MLEN = 4, ELEM_SYMB_MLEN = 2, CHARGE_MLEN = 2
};

static std::string
format(double fp_num, unsigned nint, unsigned nfrac) {
    auto maxNum = std::pow(10, nint);
    if (fp_num >= maxNum)
        throw std::invalid_argument(std::string("Number is too big (max ")
                                    + std::to_string(maxNum) + std::string(")"));

    fp_num = std::ceil(fp_num * maxNum) / static_cast<double>(maxNum);

    std::stringstream fp_num_ss;
    fp_num_ss.precision(nfrac);
    fp_num_ss.setf(std::ios::fixed, std::ios::floatfield);
    fp_num_ss << fp_num;

    return fp_num_ss.str();
}

constexpr auto COORD_MINT = 4;
constexpr auto COORD_MFRAC = 3;

constexpr auto OCCTEMP_MINT = 3;
constexpr auto OCCTEMP_MFRAC = 2;

static std::string
fcoord (double coord) {
    return format(coord, COORD_MINT, COORD_MFRAC);
}

static std::string
focctemp (double occtemp) {
    return format(occtemp, OCCTEMP_MINT, OCCTEMP_MFRAC);
}

enum direction{left, right};

static std::string check_fill (std::string val, size_t len, direction align) {
    auto val_len = val.size();
    if (val_len == 0)
        for (auto i = len; i > 0; i--, val += " ");
    else if (val_len > len)
        throw std::invalid_argument("Invalid argument length (too long): expected " +
                                    std::to_string(len) + ", got " + std::to_string(val_len));
    else {
        std::string xfix;
        for (auto i = val.size(); i < len; i++, xfix += " ");
        val = (align == right) ? xfix + val : val + xfix;
    }
    return val;
}

static std::string
check_fill(std::string val, int len) {
    return check_fill(val, len, left);
}

static void
export_to_pdb ( std::string fn,             // output filename with extension
                std::string type,           // 1-6
                std::string sn,             // 7-11  right
                std::string name,           // 13-16
                std::string alt_loc_ind,    // 17
                std::string res_name,       // 18-20 right
                std::string chain_ind,      // 22
                std::string res_seq_num,    // 23-26 right
                std::string res_ins_code,   // 27
                std::string x,              // 31-38 right
                std::string y,              // 39-46 right
                std::string z,              // 47-54 right
                std::string occ,            // 55-60 right
                std::string temp_factor,    // 61-66 right
                std::string seg_id,         // 73-76
                std::string elem_symb,      // 77-78 right
                std::string charge          // 79-80
              ){

    type = check_fill(type, TYPE_MLEN);
    sn = check_fill(sn, SN_MLEN, right);
    name = check_fill(name, NAME_MLEN);
    alt_loc_ind = check_fill(alt_loc_ind, ALT_LOC_IND_MLEN);
    res_name = check_fill(res_name, RES_NAME_MLEN, right);
    chain_ind = check_fill(chain_ind, CHAIN_IND_MLEN);
    res_seq_num = check_fill(res_seq_num, RES_SEQ_NUM_MLEN, right);
    res_ins_code = check_fill(res_ins_code, RES_INS_CODE_MLEN);
    x = check_fill(x, X_MLEN, right);
    y = check_fill(y, Y_MLEN, right);
    z = check_fill(z, Z_MLEN, right);
    occ = check_fill(occ, OCC_MLEN, right);
    temp_factor = check_fill(temp_factor, TEMP_FACTOR_MLEN, right);
    seg_id = check_fill(seg_id, SEG_ID_MLEN);
    elem_symb = check_fill(elem_symb, ELEM_SYMB_MLEN, right);
    charge = check_fill(charge, CHARGE_MLEN);

    std::ofstream pdb_file(fn, std::ofstream::app);
    pdb_file << type << sn << " " << name << alt_loc_ind << res_name << " " << chain_ind
             << res_seq_num << res_ins_code << "   " << x << y << z << occ << temp_factor
             << "     " << elem_symb << charge << std::endl;
    pdb_file.close();
}

void Grid::export_to_pdb(std::string fn) {
    remove(fn.data());
    unsigned serial_num = 1;
    for (auto particle : particles) {

        std::string sn_str = std::to_string(serial_num);

        const std::string particle_type = "ATOM";
        const std::string atom_name = "C";
        const std::string sort_of_elem = std::to_string(1);
        const std::string temp_factor = focctemp(0);

        ::export_to_pdb(fn, particle_type, std::to_string(serial_num), atom_name, "", "", "", sort_of_elem, "",
                fcoord(particle.x), fcoord(particle.y), fcoord(particle.z),
                focctemp(particle.sigma), temp_factor, "", "", "");
        serial_num++;
    }
}

/*
 * Expects that constructor has already been called, number of cells per dimention and grid size
 * are set
 */
void Grid::import_from_pdb(std::string fn) {
    std::ifstream pdb_file(fn);
    std::string line;
    while (std::getline(pdb_file, line)) {
        if (line.substr(0, 4) == "ATOM") {
            std::string x_str = line.substr(30, 8);
            std::string y_str = line.substr(38, 8);
            std::string z_str = line.substr(46, 8);
            std::string occ_str = line.substr(54, 6);

            double x = std::stod(x_str);
            double y = std::stod(y_str);
            double z = std::stod(z_str);
            double occ = std::stod(occ_str);

            particles.push_back(Particle(x, y, z, occ));
        }
    }
    pdb_file.close();

    if (particles.size() > n)
        throw std::invalid_argument("Too many particles in PDB file.\
                Either grid is badly preconfigured or PDB file is corrupted.");

    std::vector<Particle> sorted_particles;
    sorted_particles.reserve(particles.size());
    for (auto particle : particles)
        sorted_particles.push_back(particle);

    std::sort(sorted_particles.begin(), sorted_particles.end(), [](const Particle &a, const Particle &b) {
        return a.x < b.x || (a.x == b.x && a.y < b.y) || (a.x == b.x && a.y == b.y && a.z < b.z);
    });

    particles_ordered.set_data(sorted_particles.data(), sorted_particles.size());
}
