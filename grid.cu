#include "hip/hip_runtime.h"
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++, C#, and Java: http://www.viva64.com

#include <fstream>
#include <sstream>
#include <stdexcept>
#include <string>
#include <cmath>
#include <vector>
#include <iostream>

#include "grid.cuh"
#include "particle.cuh"
#include "time_measurement.cuh"

//std::random_device rd;
//std::mt19937 gen(rd());
std::mt19937 gen(1);

double random_double(double from, double to) {
    std::uniform_real_distribution<> dist(from, to);
    return dist(gen);
}

int random_int(int from, int to) {
    std::uniform_int_distribution<> dist(from, to);
    return dist(gen);
}

std::vector<Particle> Grid::get_particles() const {
    return particles;
}

double Grid::volume() const {
    return L.x * L.y * L.z;
}

size_t Grid::de_facto_n() const {
    return particles.size();
}

double Grid::density() const {
    return n / volume();
}

double Grid::packing_fraction() const {
    return (n*M_PI*pow(p_sigma, 3)) / (6.0*volume());
}

void Grid::print_grid_info() const {
    std::cout << "Simulation box size:\t\t" << L.x << " x " << L.y << " x " << L.z
        << " (volume = " << volume() << ")"<< std::endl;
    std::cout << "Num of cells per dimention:\t"
        << dim_cells.x << ", " << dim_cells.y << ", " << dim_cells.z << std::endl;
    std::cout << "Cell size:\t\t\t"
        << cell_size.x << " x " << cell_size.y << " x " << cell_size.z << std::endl;
    std::cout << "Packing fraction:\t\t" << packing_fraction() << std::endl;
    std::cout << "Density:\t\t\t" << density() << std::endl;
    std::cout << "Expected number of particles:\t" << n << std::endl;
    std::cout << "Particle's sigma (diameter):\t" << p_sigma << std::endl << std::endl;
}

template <typename T>
D3<T> Grid::normalize(const D3<T> p) const {
    D3<double> new_p = p;

    if (p.x < 0)
        new_p.x = p.x + L.x;
    if (p.y < 0)
        new_p.y = p.y + L.y;
    if (p.z < 0)
        new_p.z = p.z + L.z;
    if (p.x >= L.x)
        new_p.x = p.x - L.x;
    if (p.y >= L.y)
        new_p.y = p.y - L.y;
    if (p.z >= L.z)
        new_p.z = p.z - L.z;

    return new_p;
}

template <typename T>
D3<int> Grid::get_cell(D3<T> p) const {
    D3<double> new_p = normalize<double>(p.toD3double());

    int c_x = static_cast<int>(floor( (new_p.x / L.x) * dim_cells.x) );
    int c_y = static_cast<int>(floor( (new_p.y / L.y) * dim_cells.y) );
    int c_z = static_cast<int>(floor( (new_p.z / L.z) * dim_cells.z) );
    D3<int> cell{c_x, c_y, c_z};
    return cell;
}

template <typename T>
size_t Grid::cell_id(D3<T> p) const {
    return p.x + p.y*dim_cells.y + p.z*dim_cells.z*dim_cells.z;
}


__device__ double device_min(double a, double b) {
    return a < b ? a : b;
}

std::vector<size_t>
Grid::check_intersect_cpu(Particle particle) {
    std::vector<size_t> res;
    for (Particle p: particles) {
        auto xd = fabs(particle.x - p.x) < L.x - fabs(particle.x - p.x) ?
                        fabs(particle.x - p.x) : L.x - fabs(particle.x - p.x);

        auto yd = fabs(particle.y - p.y) < L.y - fabs(particle.y - p.y) ?
                        fabs(particle.y - p.y) : L.y - fabs(particle.y - p.y);

        auto zd = fabs(particle.z - p.z) < L.z - fabs(particle.z - p.z) ?
                        fabs(particle.z - p.z) : L.z - fabs(particle.z - p.z);

        double dist = hypot(hypot(xd, yd), zd);
        auto this_cell_id = cell_id(get_cell(p.get_coord()));
        if (dist < particle.sigma)
            res.push_back(p.id);
    }
    return res;
}

/*
 * Useful for debug purposes only, when check_intersect on CUDA is no working correctly
 */
std::vector<size_t>
Grid::check_intersect_cpu(Particle particle, uint req_cell_id) {
    std::vector<size_t> res;
    for (Particle p: particles) {
        auto xd = fabs(particle.x - p.x) < L.x - fabs(particle.x - p.x) ?
                        fabs(particle.x - p.x) : L.x - fabs(particle.x - p.x);

        auto yd = fabs(particle.y - p.y) < L.y - fabs(particle.y - p.y) ?
                        fabs(particle.y - p.y) : L.y - fabs(particle.y - p.y);

        auto zd = fabs(particle.z - p.z) < L.z - fabs(particle.z - p.z) ?
                        fabs(particle.z - p.z) : L.z - fabs(particle.z - p.z);

        double dist = hypot(hypot(xd, yd), zd);
        auto this_cell_id = cell_id(get_cell(p.get_coord()));
        if (dist < particle.sigma && this_cell_id == req_cell_id)
            res.push_back(p.id);
    }
    return res;
}

/*
 * Yet another oversload of check_intersect_cpu that accepts particle_id and ignores check for
 *  intersect with that particle. Useful in move() method
 */
std::vector<size_t>
Grid::check_intersect_cpu(Particle particle, uint req_cell_id, uint particle_id) {
    std::vector<size_t> res;
    for (Particle p: particles) {
        auto xd = fabs(particle.x - p.x) < L.x - fabs(particle.x - p.x) ?
                        fabs(particle.x - p.x) : L.x - fabs(particle.x - p.x);

        auto yd = fabs(particle.y - p.y) < L.y - fabs(particle.y - p.y) ?
                        fabs(particle.y - p.y) : L.y - fabs(particle.y - p.y);

        auto zd = fabs(particle.z - p.z) < L.z - fabs(particle.z - p.z) ?
                        fabs(particle.z - p.z) : L.z - fabs(particle.z - p.z);

        double dist = hypot(hypot(xd, yd), zd);
        auto this_cell_id = cell_id(get_cell(p.get_coord()));
        if (dist < particle.sigma && this_cell_id == req_cell_id && p.id != particle_id)
            res.push_back(p.id);
    }
    return res;
}

__global__ void update_kernel(uint *cellStartIdx, size_t cell_idx, size_t N) {
    size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId > N)
        return;
    cellStartIdx[cell_idx + threadId]++;
}

/* TODO: Rewrite with __shared__ uint* array and using parallel summing (reduce) algorithm.
 *  It should be faster like that then atomicAdd.
 */
__global__ void
check_intersect (
        const Particle *particle,
        const Particle *ordered_particles,
        const uint *cellStartIdx,
        uint curr_cell_id,
        const D3<double> *L,
        int *intersects) {

    uint startIdx = cellStartIdx[curr_cell_id];
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    auto xd = device_min( fabs(particle->x - ordered_particles[startIdx+idx].x),
                        L->x - fabs(particle->x - ordered_particles[startIdx+idx].x) );

    auto yd = device_min( fabs(particle->y - ordered_particles[startIdx+idx].y),
                        L->y - fabs(particle->y - ordered_particles[startIdx+idx].y) );

    auto zd = device_min( fabs(particle->z - ordered_particles[startIdx+idx].z),
                        L->z - fabs(particle->z - ordered_particles[startIdx+idx].z) );

    auto dist = hypot(hypot(xd, yd), zd);
    if (dist < particle->sigma)
        atomicAdd(intersects, 1);
}


__global__ void energy_single_kernel(double* energy, const Particle* particle,
                                     const Particle *particles, const uint *cellStartIdx, uint curr_cell_id,
                                     const D3<double> *L, uint curr_part_id, size_t partInCell, size_t arr_size) {

    extern __shared__ double part_energy[];

    const double sqe = -1.0;
    const double sqw = 0.2;
    const double inf = 0x7f800000;

    uint startIdx = cellStartIdx[curr_cell_id];
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    auto xd = device_min( fabs(particle->x - particles[startIdx+idx].x),
                          L->x - fabs(particle->x - particles[startIdx+idx].x) );

    auto yd = device_min( fabs(particle->y - particles[startIdx+idx].y),
                          L->y - fabs(particle->y - particles[startIdx+idx].y) );

    auto zd = device_min( fabs(particle->z - particles[startIdx+idx].z),
                          L->z - fabs(particle->z - particles[startIdx+idx].z) );

    auto dist = hypot(hypot(xd, yd), zd);

    if ((dist >= particle->sigma) && (dist < particle->sigma + sqw))
        part_energy[idx] = sqe;
    else if (dist < particle->sigma) {
        if (curr_part_id == particles[startIdx+idx].id)
            part_energy[idx] = 0.0;
        else {
            part_energy[idx] = inf;
//            printf("Error, intersected. %lu with %lu (cell %i) -- dist = %f\n",
//                   particle->id, particles[startIdx+idx].id, curr_cell_id, dist);
        }
    }
    else
        part_energy[idx] = 0;

    __syncthreads();

    if (idx+partInCell < arr_size)
        part_energy[idx+partInCell] = 0;

    for (auto i = arr_size/2; i > 0; i/=2) {
        if (idx < i)
            part_energy[idx] += part_energy[idx + i];
        __syncthreads();
    }

    if (idx == 0)
        *energy = part_energy[0];
}


void Grid::fill() {
    size_t count_tries = 0;
    size_t max_tries = 10000 * n;

    while ((particles.size() < n) && count_tries < max_tries) {

        double x = L.x * random_double(0, 1);
        double y = L.y * random_double(0, 1);
        double z = L.z * random_double(0, 1);

        Particle particle = Particle(x, y, z, p_sigma);

        Particle *cuda_particle;
        hipMalloc(&cuda_particle, sizeof(Particle));
        hipMemcpy(cuda_particle, &particle, sizeof(Particle), hipMemcpyHostToDevice);

        D3<double> p_point = particle.get_coord();
        D3<int> p_cell = get_cell(p_point);

        bool intersected = false;

        for (auto z_off = -1; z_off <= 1; ++z_off) {
            for (auto y_off = -1; y_off <= 1; ++y_off) {
                for (auto x_off = -1; x_off <= 1; ++x_off) {
                    hipMemset(intersectsCuda, 0, sizeof(int));

                    // TODO: do it better, without double
                    D3<double> offset = {x_off*cell_size.x, y_off*cell_size.y, z_off*cell_size.z};
                    uint curr_cell_id = cell_id(get_cell(p_point + offset));
                    /************************************/

                    // number of particles in cell
                    size_t partInCell = partPerCell[curr_cell_id];

                    if (partInCell == 0)
                        continue;

                    const Particle *cuda_ordered_particles = particles_ordered.get_array();

                    size_t threadsPerBlock = std::min(partInCell, MAX_BLOCK_THREADS);
                    size_t numBlocks = (partInCell + threadsPerBlock - 1) / threadsPerBlock;
                    check_intersect<<<numBlocks, threadsPerBlock>>>(cuda_particle,
                                                cuda_ordered_particles, cellStartIdx,
                                                curr_cell_id, cudaL, intersectsCuda);

                    int *intersects = new int;
                    hipMemcpy(intersects, intersectsCuda, sizeof(int),
                                                            hipMemcpyDeviceToHost);

                    if (*intersects > 0) {
                        intersected = true;
                        delete intersects;
                        break;
                    }

                    delete intersects;
                }
                if (intersected) break;
            }
            if (intersected) break;
        }

        if (!intersected) {
            complex_insert(particle);
            if (particle.id % 1000 == 0)
                std::cout << "Inserting " << particle.id << "'s" << std::endl;
        }
        else // If a particle wasn't inserted, do not increment Particle's nextId counter
            Particle::nextId--;

        count_tries++;
        hipFree(cuda_particle);
    }
    if (n != de_facto_n())
        throw std::runtime_error("Actual number of particles <de_facto_n()> in grid\
                is not equal to desired number of particles <n> after fill");

    std::cout << "Tries: " << count_tries << std::endl;

    std::cout << std::endl;
}

__global__ void backward_move_kernel(uint *cellStartIdx, size_t new_cell_id, size_t N) {
    size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId >= N)
        return;
    cellStartIdx[new_cell_id+1 + threadId]++;
}

__global__ void forward_move_kernel(uint *cellStartIdx, size_t init_cell_id, size_t N) {
    size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId >= N)
        return;
    cellStartIdx[init_cell_id+1 + threadId]--;
}

/*
 * Overload of check_intersect that accepts another argument <curr_part_id>,
 *  to ignore checking with particle with that id. This overload is used in move function
 */
__global__ void
check_intersect (
        const Particle *particle,
        const Particle *ordered_particles,
        const uint *cellStartIdx,
        uint curr_cell_id,
        const D3<double> *L,
        int *intersects,
        uint curr_part_id) {

    uint startIdx = cellStartIdx[curr_cell_id];
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (curr_part_id == ordered_particles[startIdx+idx].id)
        return;

    auto xd = device_min( fabs(particle->x - ordered_particles[startIdx+idx].x),
                        L->x - fabs(particle->x - ordered_particles[startIdx+idx].x) );

    auto yd = device_min( fabs(particle->y - ordered_particles[startIdx+idx].y),
                        L->y - fabs(particle->y - ordered_particles[startIdx+idx].y) );

    auto zd = device_min( fabs(particle->z - ordered_particles[startIdx+idx].z),
                        L->z - fabs(particle->z - ordered_particles[startIdx+idx].z) );

    auto dist = hypot(hypot(xd, yd), zd);
    if (dist < particle->sigma)
        atomicAdd(intersects, 1);
}

void Grid::move(double dispmax) {
    uint success = 0;

    for (size_t j = 0; j < n; j++) {
        auto &i = particles[random_int(0, n-1)];

        auto curr_part_id = i.id;

        D3<int> init_p_cell = get_cell(i.get_coord());
        size_t init_p_cell_id = cell_id(init_p_cell);

        double new_x = i.x + random_double(-1, 1);
        double new_y = i.y + random_double(-1, 1);
        double new_z = i.z + random_double(-1, 1);

        double vec_x = new_x - i.x;
        double vec_y = new_y - i.y;
        double vec_z = new_z - i.z;

        double vec_length = sqrt(pow(vec_x, 2) + pow(vec_y, 2) + pow(vec_z, 2));

        vec_x = vec_x / vec_length;
        vec_y = vec_y / vec_length;
        vec_z = vec_z / vec_length;

        double x = i.x + vec_x * dispmax;
        double y = i.y + vec_y * dispmax;
        double z = i.z + vec_z * dispmax;

        Particle particle = Particle(x, y, z, p_sigma);

        Particle *init_particle;
        hipMalloc(&init_particle, sizeof(Particle));
        hipMemcpy(init_particle, &i, sizeof(Particle), hipMemcpyHostToDevice);

        Particle *cuda_particle;
        hipMalloc(&cuda_particle, sizeof(Particle));
        hipMemcpy(cuda_particle, &particle, sizeof(Particle), hipMemcpyHostToDevice);

        D3<double> p_point = particle.get_coord();
        D3<int> p_cell = get_cell(p_point);
        size_t new_p_cell_id = cell_id(p_cell);

        bool intersected = false;
        bool accept = false;

        double init_en_total = 0;
        double new_en_total = 0;
        for (auto z_off = -1; z_off <= 1; ++z_off) {
            for (auto y_off = -1; y_off <= 1; ++y_off) {
                for (auto x_off = -1; x_off <= 1; ++x_off) {
                    hipMemset(intersectsCuda, 0, sizeof(int));

                    // TODO: do it better, without double
                    D3<double> offset = {x_off*cell_size.x, y_off*cell_size.y, z_off*cell_size.z};
                    uint curr_cell_id = cell_id(get_cell(p_point + offset));
                    /************************************/

                    // number of particles in cell
                    size_t partInCell = partPerCell[curr_cell_id];

                    if (partInCell == 0)
                        continue;

                    const Particle *cuda_ordered_particles = particles_ordered.get_array();


                    size_t arr_size = pow(2, ceil(log2(partInCell)));
                    energy_single_kernel<<<1, partInCell, arr_size*sizeof(double)>>>(energyCuda,
                                                                                     init_particle, cuda_ordered_particles, cellStartIdx, curr_cell_id,
                                                                                     cudaL, curr_part_id, partInCell, arr_size);

                    auto* init_en = new double;
                    hipMemcpy(init_en, energyCuda, sizeof(double), hipMemcpyDeviceToHost);
                    hipMemset(energyCuda, 0, sizeof(double));

                    energy_single_kernel<<<1, partInCell, arr_size*sizeof(double)>>>(energyCuda,
                                                                                     cuda_particle, cuda_ordered_particles, cellStartIdx, curr_cell_id,
                                                                                     cudaL, curr_part_id, partInCell, arr_size);

                    init_en_total += *init_en;

                    auto* en = new double;
                    hipMemcpy(en, energyCuda, sizeof(double), hipMemcpyDeviceToHost);
                    hipMemset(energyCuda, 0, sizeof(double));

                    if (*en > 0) {
                        intersected = true;
                        break;
                    }

                    new_en_total += *en;
                }
                if (intersected) break;
            }
            if (intersected) break;
        }

        auto delta_en = init_en_total - new_en_total;
        if (delta_en < 0) {
            accept = true;
        } else {
            if ((double) rand() / RAND_MAX < exp(-beta * delta_en))
                accept = true;
        }

        if (!intersected && accept) {
            i.x = particle.x;
            i.y = particle.y;
            i.z = particle.z;

            if (new_p_cell_id == init_p_cell_id)
                particles_ordered.update_particle(i.id, i);
            else {
                // Cell start index in ordered array for the current particle (which is inserted)
                uint *partCellStartIdx = new uint;
                hipMemcpy(partCellStartIdx, &cellStartIdx[new_p_cell_id], sizeof(uint),
                           hipMemcpyDeviceToHost);

                partPerCell[new_p_cell_id]++;
                partPerCell[init_p_cell_id]--;

                int remove_status = particles_ordered.remove_by_id(i.id);
                if (remove_status)
                    throw std::runtime_error("Error in remove");

                int insert_status = particles_ordered.insert(i, *partCellStartIdx);
                if (insert_status)
                    throw std::runtime_error("Error in insert");

                size_t cells_in_range = init_p_cell_id > new_p_cell_id ?
                            init_p_cell_id - new_p_cell_id : new_p_cell_id - init_p_cell_id;

                size_t threadsPerBlock = std::min(cells_in_range, MAX_BLOCK_THREADS);
                size_t numBlocks = (cells_in_range + threadsPerBlock - 1) / threadsPerBlock;

                if (init_p_cell_id > new_p_cell_id)
                    backward_move_kernel<<<numBlocks, threadsPerBlock>>>
                                (cellStartIdx, new_p_cell_id, cells_in_range);

                else if (init_p_cell_id < new_p_cell_id)
                    forward_move_kernel<<<numBlocks, threadsPerBlock>>>
                                (cellStartIdx, init_p_cell_id, cells_in_range);
            }
            success++;
        }

        hipFree(cuda_particle);
    }
    std::cout << success << " moved" << std::endl;
}

void Grid::system_energy() {
    energy = 0;

    for (auto &particle: particles) {
        auto curr_part_id = particle.id;
        D3<double> p_point = particle.get_coord();

        Particle *cuda_particle;
        hipMalloc(&cuda_particle, sizeof(Particle));
        hipMemcpy(cuda_particle, &particle, sizeof(Particle), hipMemcpyHostToDevice);

        const Particle *cuda_ordered_particles = particles_ordered.get_array();
        for (auto z_off = -1; z_off <= 1; ++z_off) {
            for (auto y_off = -1; y_off <= 1; ++y_off) {
                for (auto x_off = -1; x_off <= 1; ++x_off) {
                    D3<double> offset = {x_off*cell_size.x, y_off*cell_size.y, z_off*cell_size.z};
                    uint curr_cell_id = cell_id(get_cell(p_point + offset));

                    size_t partInCell = partPerCell[curr_cell_id];

                    if (partInCell == 0)
                        continue;
                    size_t arr_size = pow(2, ceil(log2(partInCell)));
                    energy_single_kernel<<<1, partInCell, arr_size*sizeof(double)>>>(energyCuda,
                            cuda_particle, cuda_ordered_particles, cellStartIdx, curr_cell_id,
                            cudaL, curr_part_id, partInCell, arr_size);

                    auto* en = new double;
                    hipMemcpy(en, energyCuda, sizeof(double), hipMemcpyDeviceToHost);
                    hipMemset(energyCuda, 0, sizeof(double));

                    energy += *en;
                    delete en;
                }
            }
        }
    }
}


enum paramsMLen{
    TYPE_MLEN = 6, SN_MLEN = 5, NAME_MLEN = 4, ALT_LOC_IND_MLEN = 1, RES_NAME_MLEN = 3,
    CHAIN_IND_MLEN = 1, RES_SEQ_NUM_MLEN = 4, RES_INS_CODE_MLEN = 1,
    X_MLEN = 8, Y_MLEN = 8, Z_MLEN = 8, OCC_MLEN = 6, TEMP_FACTOR_MLEN = 6,
    SEG_ID_MLEN = 4, ELEM_SYMB_MLEN = 2, CHARGE_MLEN = 2
};

static std::string
format(double fp_num, unsigned nint, unsigned nfrac) {
    auto maxNum = std::pow(10, nint);
    if (fp_num >= maxNum)
        throw std::invalid_argument(std::string("Number is too big (max ")
                                    + std::to_string(maxNum) + std::string(")"));

    fp_num = std::ceil(fp_num * maxNum) / static_cast<double>(maxNum);

    std::stringstream fp_num_ss;
    fp_num_ss.precision(nfrac);
    fp_num_ss.setf(std::ios::fixed, std::ios::floatfield);
    fp_num_ss << fp_num;

    return fp_num_ss.str();
}

constexpr auto COORD_MINT = 4;
constexpr auto COORD_MFRAC = 3;

constexpr auto OCCTEMP_MINT = 3;
constexpr auto OCCTEMP_MFRAC = 2;

static std::string
fcoord (double coord) {
    return format(coord, COORD_MINT, COORD_MFRAC);
}

static std::string
focctemp (double occtemp) {
    return format(occtemp, OCCTEMP_MINT, OCCTEMP_MFRAC);
}

enum direction{left, right};

static std::string check_fill (std::string val, size_t len, direction align) {
    auto val_len = val.size();
    if (val_len == 0)
        for (auto i = len; i > 0; i--, val += " ");
    else if (val_len > len)
        throw std::invalid_argument("Invalid argument length (too long): expected " +
                                    std::to_string(len) + ", got " + std::to_string(val_len));
    else {
        std::string xfix;
        for (auto i = val.size(); i < len; i++, xfix += " ");
        val = (align == right) ? xfix + val : val + xfix;
    }
    return val;
}

static std::string
check_fill(std::string val, int len) {
    return check_fill(std::move(val), len, left);
}

static void
export_to_pdb ( const std::string& fn,             // output filename with extension
                std::string type,           // 1-6
                std::string sn,             // 7-11  right
                std::string name,           // 13-16
                std::string alt_loc_ind,    // 17
                std::string res_name,       // 18-20 right
                std::string chain_ind,      // 22
                std::string res_seq_num,    // 23-26 right
                std::string res_ins_code,   // 27
                std::string x,              // 31-38 right
                std::string y,              // 39-46 right
                std::string z,              // 47-54 right
                std::string occ,            // 55-60 right
                std::string temp_factor,    // 61-66 right
                std::string seg_id,         // 73-76
                std::string elem_symb,      // 77-78 right
                std::string charge          // 79-80
              ){

    type = check_fill(type, TYPE_MLEN);
    sn = check_fill(sn, SN_MLEN, right);
    name = check_fill(name, NAME_MLEN);
    alt_loc_ind = check_fill(alt_loc_ind, ALT_LOC_IND_MLEN);
    res_name = check_fill(res_name, RES_NAME_MLEN, right);
    chain_ind = check_fill(chain_ind, CHAIN_IND_MLEN);
    res_seq_num = check_fill(res_seq_num, RES_SEQ_NUM_MLEN, right);
    res_ins_code = check_fill(res_ins_code, RES_INS_CODE_MLEN);
    x = check_fill(x, X_MLEN, right);
    y = check_fill(y, Y_MLEN, right);
    z = check_fill(z, Z_MLEN, right);
    occ = check_fill(occ, OCC_MLEN, right);
    temp_factor = check_fill(temp_factor, TEMP_FACTOR_MLEN, right);
    seg_id = check_fill(seg_id, SEG_ID_MLEN);
    elem_symb = check_fill(elem_symb, ELEM_SYMB_MLEN, right);
    charge = check_fill(charge, CHARGE_MLEN);

    std::ofstream pdb_file(fn, std::ofstream::app);
    pdb_file << type << sn << " " << name << alt_loc_ind << res_name << " " << chain_ind
             << res_seq_num << res_ins_code << "   " << x << y << z << occ << temp_factor
             << "     " << elem_symb << charge << std::endl;
    pdb_file.close();
}

void Grid::export_to_pdb(const std::string& fn) {
    remove(fn.data());
    unsigned serial_num = 1;
    for (auto particle : particles) {

        std::string sn_str = std::to_string(serial_num);

        const std::string particle_type = "ATOM";
        const std::string atom_name = "C";
        const std::string sort_of_elem = std::to_string(1);
        const std::string temp_factor = focctemp(0);

        ::export_to_pdb(fn, particle_type, std::to_string(serial_num), atom_name, "", "", "",
                sort_of_elem, "", fcoord(particle.x), fcoord(particle.y), fcoord(particle.z),
                focctemp(particle.sigma), temp_factor, "", "", "");
        serial_num++;
    }
}

void Grid::export_to_cf(const std::string& fn) {
    std::ofstream cf_file(fn);
    if (!cf_file)
        throw std::runtime_error("Error while opening file for export " + fn);

    for (auto p: particles) {
        char buff[256];
        sprintf(buff, "%15ld%15ld%4d%20.10lf%20.10lf%20.10lf\n", p.id, p.id, 1, p.x, p.y, p.z);
        std::string buff_str{buff};
        cf_file << buff_str;
    }
}

void Grid::import_from_cf(const std::string& fn) {
    // import from custom format file
    std::ifstream cf_file(fn);
    if (!cf_file)
        throw std::runtime_error("Error while opening file for import " + fn);

    std::string line;

    while (std::getline(cf_file, line)) {
        std::stringstream ss(line);

        // skip first 3 columns, because they are useless for particle constructor
        long trash;
        ss >> trash >> trash >> trash;

        double x, y, z;
        ss >> x >> y >> z;
        Particle p(x, y, z, p_sigma);
        complex_insert(p);
    }
    cf_file.close();

    if (particles.size() != n)
        throw std::invalid_argument("During import: too many particles in CF file.\
                Either grid is badly preconfigured or CF file is corrupted.");
}

void Grid::complex_insert(Particle p) {
    particles.push_back(p);
    auto p_cell_id = cell_id(get_cell(p.get_coord()));

    // Cell start index in ordered array for the current particle (which is inserted)
    uint *partCellStartIdx = new uint;
    hipMemcpy(partCellStartIdx, &cellStartIdx[p_cell_id], sizeof(uint),
                                                hipMemcpyDeviceToHost);

    particles_ordered.insert(p, *partCellStartIdx);
    partPerCell[p_cell_id]++;

    if (n_cells < p_cell_id + 1)
        throw std::runtime_error("Cell_idx > number of cells, which is impossible");

    size_t N = n_cells-p_cell_id-1;
    if (N > 0) {
        size_t threadsPerBlock = std::min(N, MAX_BLOCK_THREADS);
        size_t numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;
        update_kernel<<<numBlocks, threadsPerBlock>>>(cellStartIdx, p_cell_id+1, N);
    }

    free(partCellStartIdx);
}
