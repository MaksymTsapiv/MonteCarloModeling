#include "hip/hip_runtime.h"
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++, C#, and Java: http://www.viva64.com

#include <fstream>
#include <sstream>
#include <stdexcept>
#include <string>
#include <cmath>
#include <vector>
#include <iostream>
#include <map>
#include <stack>
#include <algorithm>

#include "grid.cuh"
#include "particle.cuh"
#include "time_measurement.cuh"

//std::random_device rd;
//std::mt19937 gen(rd());
std::mt19937 gen(1);

double random_double(double from, double to) {
    std::uniform_real_distribution<> dist(from, to);
    return dist(gen);
}

int random_int(int from, int to) {
    std::uniform_int_distribution<> dist(from, to);
    return dist(gen);
}

std::vector<Particle> Grid::get_particles() const {
    return particles;
}

double Grid::volume() const {
    return L.x * L.y * L.z;
}

size_t Grid::de_facto_n() const {
    return particles.size();
}

double Grid::density() const {
    return n / volume();
}

double Grid::packing_fraction() const {
    return (n*M_PI*pow(pSigma, 3)) / (6.0*volume());
}

void Grid::print_grid_info() const {
    std::cout << "Simulation box size:\t\t" << L.x << " x " << L.y << " x " << L.z
        << " (volume = " << volume() << ")"<< std::endl;
    std::cout << "Num of cells per dimention:\t"
        << dimCells.x << " x " << dimCells.y << " x " << dimCells.z << "  = "
        << n_cells << std::endl;
    std::cout << "Cell size:\t\t\t"
        << cellSize.x << " x " << cellSize.y << " x " << cellSize.z << std::endl;
    std::cout << "Average particles per cell:\t" << static_cast<double>(n)/n_cells << std::endl;
    std::cout << "Max particles per cell:\t\t" << maxPartPerCell << std::endl;
    std::cout << "Packing fraction:\t\t" << packing_fraction() << std::endl;
    std::cout << "Density:\t\t\t" << density() << std::endl;
    std::cout << "Temperature:\t\t\t" << temp << std::endl;
    std::cout << "Expected number of particles:\t" << n << std::endl;
    std::cout << "Particle's sigma (diameter):\t" << pSigma << std::endl << std::endl;
}

template <typename T>
D3<T> Grid::normalize(const D3<T> p) const {
    D3<T> new_p = p;

    if (p.x < 0)
        new_p.x = p.x + L.x;
    if (p.y < 0)
        new_p.y = p.y + L.y;
    if (p.z < 0)
        new_p.z = p.z + L.z;
    if (p.x >= L.x)
        new_p.x = p.x - L.x;
    if (p.y >= L.y)
        new_p.y = p.y - L.y;
    if (p.z >= L.z)
        new_p.z = p.z - L.z;

    return new_p;
}

template <typename T>
D3<int> Grid::cell(D3<T> p) const {
    D3<double> new_p = normalize<double>(p.toD3double());

    int c_x = static_cast<int>(floor( (new_p.x / L.x) * dimCells.x) );
    int c_y = static_cast<int>(floor( (new_p.y / L.y) * dimCells.y) );
    int c_z = static_cast<int>(floor( (new_p.z / L.z) * dimCells.z) );
    D3<int> cell{c_x, c_y, c_z};
    return cell;
}

template <typename T>
size_t Grid::cell_id(D3<T> p) const {
    return p.x + p.y*dimCells.y + p.z*dimCells.z*dimCells.z;
}


__device__ double device_min(double a, double b) {
    return a < b ? a : b;
}

std::vector<size_t>
Grid::check_intersect_cpu(Particle particle) {
    std::vector<size_t> res;
    for (Particle p: particles) {
        auto xd = fabs(particle.x - p.x) < L.x - fabs(particle.x - p.x) ?
                        fabs(particle.x - p.x) : L.x - fabs(particle.x - p.x);

        auto yd = fabs(particle.y - p.y) < L.y - fabs(particle.y - p.y) ?
                        fabs(particle.y - p.y) : L.y - fabs(particle.y - p.y);

        auto zd = fabs(particle.z - p.z) < L.z - fabs(particle.z - p.z) ?
                        fabs(particle.z - p.z) : L.z - fabs(particle.z - p.z);

        double dist = hypot(hypot(xd, yd), zd);
        auto this_cell_id = cell_id(cell(p.get_coord()));
        if (dist < particle.sigma)
            res.push_back(p.id);
    }
    return res;
}

/*
 * Useful for debug purposes only, when check_intersect on CUDA is no working correctly
 */
std::vector<size_t>
Grid::check_intersect_cpu(Particle particle, uint req_cell_id) {
    std::vector<size_t> res;
    for (Particle p: particles) {
        auto xd = fabs(particle.x - p.x) < L.x - fabs(particle.x - p.x) ?
                        fabs(particle.x - p.x) : L.x - fabs(particle.x - p.x);

        auto yd = fabs(particle.y - p.y) < L.y - fabs(particle.y - p.y) ?
                        fabs(particle.y - p.y) : L.y - fabs(particle.y - p.y);

        auto zd = fabs(particle.z - p.z) < L.z - fabs(particle.z - p.z) ?
                        fabs(particle.z - p.z) : L.z - fabs(particle.z - p.z);

        double dist = hypot(hypot(xd, yd), zd);
        auto this_cell_id = cell_id(cell(p.get_coord()));
        if (dist < particle.sigma && this_cell_id == req_cell_id)
            res.push_back(p.id);
    }
    return res;
}

/*
 * Yet another oversload of check_intersect_cpu that accepts particle_id and ignores check for
 *  intersect with that particle. Useful in move() method
 */
std::vector<size_t>
Grid::check_intersect_cpu(Particle particle, uint req_cell_id, uint particle_id) {
    std::vector<size_t> res;
    for (Particle p: particles) {
        auto xd = fabs(particle.x - p.x) < L.x - fabs(particle.x - p.x) ?
                        fabs(particle.x - p.x) : L.x - fabs(particle.x - p.x);

        auto yd = fabs(particle.y - p.y) < L.y - fabs(particle.y - p.y) ?
                        fabs(particle.y - p.y) : L.y - fabs(particle.y - p.y);

        auto zd = fabs(particle.z - p.z) < L.z - fabs(particle.z - p.z) ?
                        fabs(particle.z - p.z) : L.z - fabs(particle.z - p.z);

        double dist = hypot(hypot(xd, yd), zd);
        auto this_cell_id = cell_id(cell(p.get_coord()));
        if (dist < particle.sigma && this_cell_id == req_cell_id && p.id != particle_id)
            res.push_back(p.id);
    }
    return res;
}

__global__ void update_kernel(uint *cellStartIdx, size_t cell_idx, size_t N) {
    size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId > N)
        return;
    cellStartIdx[cell_idx + threadId]++;
}

/* TODO: Rewrite with __shared__ uint* array and using parallel summing (reduce) algorithm.
 *  It should be faster like that then atomicAdd.
 */
__global__ void
check_intersect (
        const Particle *particle,
        const Particle *ordered_particles,
        const uint *cellStartIdx,
        uint curr_cell_id,
        const D3<double> *L,
        int *intersects) {

    uint startIdx = cellStartIdx[curr_cell_id];
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    auto xd = device_min( fabs(particle->x - ordered_particles[startIdx+idx].x),
                        L->x - fabs(particle->x - ordered_particles[startIdx+idx].x) );

    auto yd = device_min( fabs(particle->y - ordered_particles[startIdx+idx].y),
                        L->y - fabs(particle->y - ordered_particles[startIdx+idx].y) );

    auto zd = device_min( fabs(particle->z - ordered_particles[startIdx+idx].z),
                        L->z - fabs(particle->z - ordered_particles[startIdx+idx].z) );

    auto dist = hypot(hypot(xd, yd), zd);
    if (dist < particle->sigma)
        atomicAdd(intersects, 1);
}


__global__ void energy_all_cell_kernel(double* energy, Particle particle, const uint *partPerCell,
                                       const Particle *particles, const uint *cellStartIdx,
                                       const D3<double> *L,
                                       const AdjCells *adjCells, uint currPartCell)
{
    extern __shared__ double part_energy[];

    const double sqe = -1.0;
    const double sqw = 0.2;
    const double inf = 0x7f800000;

    uint blockI = blockIdx.x;
    if (blockI >= 27)
        return;

    uint currCellId = adjCells[currPartCell].ac[blockI];
    uint startIdx = cellStartIdx[currCellId];

    if (threadIdx.x < partPerCell[currCellId]) {
        auto xd = device_min( fabs(particle.x - particles[startIdx+threadIdx.x].x),
                              L->x - fabs(particle.x - particles[startIdx+threadIdx.x].x) );

        auto yd = device_min( fabs(particle.y - particles[startIdx+threadIdx.x].y),
                              L->y - fabs(particle.y - particles[startIdx+threadIdx.x].y) );

        auto zd = device_min( fabs(particle.z - particles[startIdx+threadIdx.x].z),
                              L->z - fabs(particle.z - particles[startIdx+threadIdx.x].z) );

        auto dist = hypot(hypot(xd, yd), zd);

        if ((dist >= particle.sigma) && (dist < particle.sigma + sqw))
            part_energy[threadIdx.x] = sqe;
        else if (dist < particle.sigma) {
            if (particle.id == particles[startIdx+threadIdx.x].id)
                part_energy[threadIdx.x] = 0.0;
            else {
                part_energy[threadIdx.x] = inf;
                printf("Error, intersected. %lu with %lu (cell %i) -- dist = %f\n",
                       particle.id, particles[startIdx+threadIdx.x].id, currCellId, dist);
            }
        }
        else
            part_energy[threadIdx.x] = 0;
    }
    else
        part_energy[threadIdx.x] = 0;

    __syncthreads();

    for (auto i = blockDim.x/2; i > 0; i/=2) {
        if (threadIdx.x < i)
            part_energy[threadIdx.x] += part_energy[threadIdx.x + i];
        __syncthreads();
    }

    if (threadIdx.x == 0)
        energy[blockI] = part_energy[0];

}




size_t Grid::fill() {
    size_t count_tries = 0;
    size_t max_tries = 10000 * n;

    while ((particles.size() < n) && count_tries < max_tries) {

        double x = L.x * random_double(0, 1);
        double y = L.y * random_double(0, 1);
        double z = L.z * random_double(0, 1);

        Particle particle = Particle(x, y, z, pSigma);

        auto pCellId = cell_id(cell(particle.get_coord()));

        bool intersected = false;
        double energy_loc = 0.0;

        energy_all_cell_kernel<<<nAdjCells, maxPartPerCell2pow, maxPartPerCell2pow*sizeof(double)>>>
                        (energiesCuda, particle, partPerCellCuda, orderedParticlesCuda.get_array(),
                         cellStartIdxCuda, cudaL, cnCuda, pCellId);

        auto *energies = new double[nAdjCells];
        hipMemcpy(energies, energiesCuda, sizeof(double) * nAdjCells, hipMemcpyDeviceToHost);

        for (uint i = 0; i < nAdjCells; i++)
            energy_loc += energies[i];

        if (energy_loc > 0)
            intersected = true;

        if (!intersected) {
            complex_insert(particle);
            if (particle.id % 1000 == 0)
                std::cout << "Inserting " << particle.id << "'s" << std::endl;
        }
        else // If a particle wasn't inserted, do not increment Particle's nextId counter
            Particle::nextId--;

        count_tries++;
    }
    if (n != de_facto_n())
        throw std::runtime_error("Actual number of particles <de_facto_n()> in grid\
                is not equal to desired number of particles <n> after fill");

    hipMemcpy(partPerCellCuda, partPerCell, sizeof(uint)*n_cells, hipMemcpyHostToDevice);

    return count_tries;
}



void Grid::dfs_cluster(double connectDist) {
    std::vector<int> in_cluster(static_cast<int>(n), 0);
    std::stack<size_t> pidStack;

    uint* cellStartIdx = new uint[n_cells];
    Particle* orderedParticles = new Particle[n];

    hipMemcpy(cellStartIdx, cellStartIdxCuda, sizeof(uint) * n_cells, hipMemcpyDeviceToHost);
    hipMemcpy(orderedParticles, orderedParticlesCuda.get_array(),
                sizeof(Particle) * n, hipMemcpyDeviceToHost);


    for (auto i = 0; i < n; ++i) {
        if (in_cluster[i] > 0)
            continue;
        const auto particle = particles[i];

        pidStack.push(particle.id);
        in_cluster[particle.id]++;

        while (!pidStack.empty()) {
            const auto part = particles[pidStack.top()];
            const auto parentClusterId = part.clusterId;
            pidStack.pop();

            D3<double> p_point = part.get_coord();

            for (auto z_off = -1; z_off <= 1; ++z_off) {
                for (auto y_off = -1; y_off <= 1; ++y_off) {
                    for (auto x_off = -1; x_off <= 1; ++x_off) {
                        hipMemset(intersectsCuda, 0, sizeof(int));
                        D3<double> offset = {x_off*cellSize.x, y_off*cellSize.y, z_off*cellSize.z};
                        size_t curr_cell_id = cell_id(cell(p_point + offset));

                        size_t partInCell = partPerCell[curr_cell_id];

                        if (partInCell == 0)
                            continue;

                        uint nextCellStartIdx;
                        if (curr_cell_id+1 == n_cells)
                            nextCellStartIdx = n_cells;
                        else
                            nextCellStartIdx = cellStartIdx[curr_cell_id+1];

                        for (int j = cellStartIdx[curr_cell_id]; j < nextCellStartIdx; ++j) {
                            auto &currPart = orderedParticles[j];
                            if (currPart.id == part.id)
                                continue;

                            auto xd = std::min( fabs(part.x - currPart.x),
                                    L.x - fabs(part.x - currPart.x) );

                            auto yd = std::min( fabs(part.y - currPart.y),
                                    L.y - fabs(part.y - currPart.y) );

                            auto zd = std::min( fabs(part.z - currPart.z),
                                    L.z - fabs(part.z - currPart.z) );

                            auto dist = hypot(hypot(xd, yd), zd);

                            if (dist <= connectDist && !in_cluster[currPart.id]) {
                                pidStack.push(currPart.id);
                                in_cluster[currPart.id]++;
                                orderedParticles[j].clusterId = parentClusterId;
                                particles[j].clusterId = parentClusterId;
                            }
                        }
                    }
                }
            }
        }
    }

    orderedParticlesCuda.set_data(orderedParticles, n);
}


void Grid::check_cluster() {
    std::map<size_t, size_t> clusters;
    for (auto & particle : particles)
        clusters[particle.clusterId]++;

    for (auto & cluster : clusters)
        std::cout << cluster.first << " : " << cluster.second << '\n';
}




__global__ void backward_move_kernel(uint *cellStartIdx, size_t new_cell_id, size_t N) {
    size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId >= N)
        return;
    cellStartIdx[new_cell_id+1 + threadId]++;
}

__global__ void forward_move_kernel(uint *cellStartIdx, size_t init_cell_id, size_t N) {
    size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId >= N)
        return;
    cellStartIdx[init_cell_id+1 + threadId]--;
}

/*
 * Overload of check_intersect that accepts another argument <curr_part_id>,
 *  to ignore checking with particle with that id. This overload is used in move function
 */
__global__ void
check_intersect (
        const Particle *particle,
        const Particle *ordered_particles,
        const uint *cellStartIdx,
        uint curr_cell_id,
        const D3<double> *L,
        int *intersects,
        uint curr_part_id) {

    uint startIdx = cellStartIdx[curr_cell_id];
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (curr_part_id == ordered_particles[startIdx+idx].id)
        return;

    auto xd = device_min( fabs(particle->x - ordered_particles[startIdx+idx].x),
                        L->x - fabs(particle->x - ordered_particles[startIdx+idx].x) );

    auto yd = device_min( fabs(particle->y - ordered_particles[startIdx+idx].y),
                        L->y - fabs(particle->y - ordered_particles[startIdx+idx].y) );

    auto zd = device_min( fabs(particle->z - ordered_particles[startIdx+idx].z),
                        L->z - fabs(particle->z - ordered_particles[startIdx+idx].z) );

    auto dist = hypot(hypot(xd, yd), zd);
    if (dist < particle->sigma)
        atomicAdd(intersects, 1);
}



/*
 * Update cluster id of particles1 & particles2. It is supposed to be called on all particles
 *  in the Grid. It will check if particle is in the cluster that should be changed and
 *  updates such particles's cluster id.
 */
__global__ void update_parts_cluster_kernel(Particle *particles1, Particle *particles2,
                const size_t* clusterIds, size_t nUniqueClusters, size_t newClusterId, size_t N)
{
    uint threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId >= N)
        return;

    bool returnFlag = false;
    // Check if current particle's clusterId is in clusterIds array, so that it should be changed
    for (size_t i = 0; i < nUniqueClusters; i++) {
        if (clusterIds[i] == particles1[threadId].clusterId) {
            particles1[threadId].clusterId = newClusterId;
            if (returnFlag)
                return;
            returnFlag = true;
        }
        if (clusterIds[i] == particles2[threadId].clusterId) {
            particles2[threadId].clusterId = newClusterId;
            if (returnFlag)
                return;
            returnFlag = true;
        }
    }
}



__global__ void energy_and_cluster_kernel(double* energy, Particle particle, const uint *partPerCell,
                                       const Particle *particles, const uint *cellStartIdx,
                                       const D3<double> *L, const AdjCells *adjCells,
                                       uint currPartCell, int *clusters)
{
    extern __shared__ double part_energy[];

    const double sqe = -1.0;
    const double sqw = 0.2;
    const double inf = 0x7f800000;

    uint blockI = blockIdx.x;
    if (blockI >= 27)
        return;

    uint currCellId = adjCells[currPartCell].ac[blockI];
    uint startIdx = cellStartIdx[currCellId];

    clusters[blockDim.x * blockI + threadIdx.x] = -1;

    if (threadIdx.x < partPerCell[currCellId]) {
        auto xd = device_min( fabs(particle.x - particles[startIdx+threadIdx.x].x),
                              L->x - fabs(particle.x - particles[startIdx+threadIdx.x].x) );

        auto yd = device_min( fabs(particle.y - particles[startIdx+threadIdx.x].y),
                              L->y - fabs(particle.y - particles[startIdx+threadIdx.x].y) );

        auto zd = device_min( fabs(particle.z - particles[startIdx+threadIdx.x].z),
                              L->z - fabs(particle.z - particles[startIdx+threadIdx.x].z) );

        auto dist = hypot(hypot(xd, yd), zd);

        if ((dist >= particle.sigma) && (dist < particle.sigma + sqw)) {
            part_energy[threadIdx.x] = sqe;
            clusters[blockDim.x * blockI + threadIdx.x] = particles[startIdx+threadIdx.x].clusterId;
            // printf("   clusters[%i] = %lu\n", blockDim.x * blockI + threadIdx.x, particles[startIdx+threadIdx.x].clusterId);
        }
        else if (dist < particle.sigma) {
            if (particle.id == particles[startIdx+threadIdx.x].id)
                part_energy[threadIdx.x] = 0.0;
            else {
                part_energy[threadIdx.x] = inf;
            }
        }
        else
            part_energy[threadIdx.x] = 0;
    }
    else
        part_energy[threadIdx.x] = 0;

    __syncthreads();

    for (auto i = blockDim.x/2; i > 0; i/=2) {
        if (threadIdx.x < i)
            part_energy[threadIdx.x] += part_energy[threadIdx.x + i];
        __syncthreads();
    }

    if (threadIdx.x == 0)
        energy[blockI] = part_energy[0];

}

__global__ void print_some_parts(const Particle *particles, const uint *cellStartIdx) {
    // for (int i = 0; i < 64; i++){
    //     printf("Cell %i : %u\n", i, cellStartIdx[i]);
    // }
    // printf("\n\n");
    for (int i = 0; i < 100; i++){
        auto currPart = particles[cellStartIdx[0]+i];
        printf("%i) %lu: (%f, %f, %f)\n", i, currPart.id, currPart.x, currPart.y, currPart.z);
    }
    // for (int i = 0; i < 4; i++){
    //     auto currPart = particles[cellStartIdxCuda[11]+i];
    //     printf("%lu: (%f, %f, %f)\n", currPart.id, currPart.x, currPart.y, currPart.z);
    // }
}

size_t Grid::move(double dispmax, int step) {
    uint success = 0;

    /* Contains ids of clusters of particles that interact with the given particle.
     *  It should be int because if i-th particle doesn't interact with current particle
     *  clustersIdCuda[i] = -1
     */
    int *clustersIdCuda;
    auto clustersArrSizeBytes = nAdjCells * maxPartPerCell2pow * sizeof(int);
    hipMalloc(&clustersIdCuda, clustersArrSizeBytes);

    // Capacity of shared memory. It will be used in configurations upon kernel function call
    auto sharedMemSizeBytes = maxPartPerCell2pow * sizeof(double);

    for (size_t _ = 0; _ < n; _++) {
        std::cout << "Cycle " << _ << std::endl;
        auto &currPart = particles[random_int(0, n-1)];

        D3<int> init_p_cell = cell(currPart.get_coord());
        size_t initPCellId = cell_id(init_p_cell);

        double new_x = currPart.x + random_double(-1, 1);
        double new_y = currPart.y + random_double(-1, 1);
        double new_z = currPart.z + random_double(-1, 1);

        double vec_x = new_x - currPart.x;
        double vec_y = new_y - currPart.y;
        double vec_z = new_z - currPart.z;

        double vec_length = sqrt(pow(vec_x, 2) + pow(vec_y, 2) + pow(vec_z, 2));

        vec_x = vec_x / vec_length;
        vec_y = vec_y / vec_length;
        vec_z = vec_z / vec_length;

        double x = currPart.x + vec_x * dispmax;
        double y = currPart.y + vec_y * dispmax;
        double z = currPart.z + vec_z * dispmax;

        Particle particle = Particle(x, y, z, pSigma);
        Particle::nextId--;     // Reset Particle::nextId, because <particle> is temporary particle
        particle.id = currPart.id;

        D3<double> p_point = particle.get_coord();
        size_t newPCellId = cell_id(cell(p_point));

        bool intersected = false;
        bool accept = false;

        if (_ == 78)
            auto jii = 0;

        energy_all_cell_kernel<<<nAdjCells, maxPartPerCell2pow, maxPartPerCell2pow*sizeof(double)>>>
                        (energiesCuda, currPart, partPerCellCuda, orderedParticlesCuda.get_array(),
                         cellStartIdxCuda, cudaL, cnCuda, initPCellId);

        auto *preEnergies = new double[nAdjCells];
        hipMemcpy(preEnergies, energiesCuda, sizeof(double) * nAdjCells, hipMemcpyDeviceToHost);

        double preEnergy = 0.0;
        for (uint k = 0; k < nAdjCells; k++)
            preEnergy += preEnergies[k];

        delete[] preEnergies;

        energy_and_cluster_kernel<<<nAdjCells, maxPartPerCell2pow, sharedMemSizeBytes>>>
                        (energiesCuda, particle, partPerCellCuda, orderedParticlesCuda.get_array(),
                         cellStartIdxCuda, cudaL, cnCuda, newPCellId, clustersIdCuda);

        auto *postEnergies = new double[nAdjCells];
        hipMemcpy(postEnergies, energiesCuda, sizeof(double) * nAdjCells, hipMemcpyDeviceToHost);

        double postEnergy = 0.0;
        for (uint k = 0; k < nAdjCells; k++)
            postEnergy += postEnergies[k];

        delete[] postEnergies;

        if (postEnergy > 0)
            intersected = true;

        auto delta_en = postEnergy - preEnergy;

        if (delta_en > 0) {
            if ((double) rand() / RAND_MAX < exp(-beta * delta_en))
                accept = true;
        } else {
            accept = true;
        }

        if (!intersected && accept) {
            energy += delta_en;
            currPart.x = particle.x;
            currPart.y = particle.y;
            currPart.z = particle.z;

            int *clusters = new int[nAdjCells * maxPartPerCell2pow];
            hipMemcpy(clusters, clustersIdCuda, clustersArrSizeBytes, hipMemcpyDeviceToHost);

            std::vector<size_t> uniqueClusters;

            for (int j = 0; j < nAdjCells * maxPartPerCell2pow; j++) {
                if (clusters[j] != -1)
                    if (std::find(uniqueClusters.begin(), uniqueClusters.end(), clusters[j])
                                                                        == uniqueClusters.end())
                        uniqueClusters.push_back(clusters[j]);
            }

            if (uniqueClusters.size() > 1) {
                size_t minClusterId = n;
                for (const auto &clusterId : uniqueClusters)
                    if (clusterId < minClusterId)
                        minClusterId = clusterId;

                // Remove minClusterId from uniqueClusters so that uniqueClusters
                //  contains only clusters that should be changed
                uniqueClusters.erase(std::find(uniqueClusters.begin(), uniqueClusters.end(),
                                                                                minClusterId));

                size_t threadsPerBlock = std::min(n, MAX_BLOCK_THREADS);
                size_t numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

                // uniqueClusters size in bytes
                size_t ucSizeBytes = sizeof(size_t) * uniqueClusters.size();
                size_t *uniqueClustersCuda;
                hipMalloc(&uniqueClustersCuda, ucSizeBytes);
                hipMemcpy(uniqueClustersCuda, uniqueClusters.data(), ucSizeBytes,
                                                            hipMemcpyHostToDevice);

                /* Copying from host to device & then vice versa to update particles' cluster
                 *   in both host vector Grid::particles and device array orderedParticlesCuda
                 *   is a very bad idea, it should be super slow.
                 * TODO: Redesign it.   HOW?
                 */
                Particle *particlesCuda;
                hipMalloc(&particlesCuda, n * sizeof(Particle));
                hipMemcpy(particlesCuda, particles.data(), n * sizeof(Particle),
                                                            hipMemcpyHostToDevice);

                update_parts_cluster_kernel<<<numBlocks, threadsPerBlock>>>
                            (orderedParticlesCuda.get_mutable_array(), particlesCuda, uniqueClustersCuda,
                             uniqueClusters.size(), minClusterId, n);

                Particle *particlesTmp = new Particle[n];
                hipMemcpy(particlesTmp, particlesCuda, n * sizeof(Particle), hipMemcpyDeviceToHost);
                particles = std::vector<Particle>{particlesTmp, particlesTmp + n};
            }

            if (newPCellId == initPCellId) {
                auto updateStatus = orderedParticlesCuda.update_particle(currPart.id, currPart);
                if (updateStatus)
                    throw std::runtime_error("Error in update_particle");
            }

            else {
                // Cell start index in ordered array for the current particle (which is inserted)
                uint *partCellStartIdx = new uint;
                hipMemcpy(partCellStartIdx, &cellStartIdxCuda[newPCellId], sizeof(uint),
                           hipMemcpyDeviceToHost);

                partPerCell[newPCellId]++;
                partPerCell[initPCellId]--;

                hipMemcpy(&partPerCellCuda[newPCellId], &partPerCell[newPCellId],
                                                    sizeof(uint), hipMemcpyHostToDevice);
                hipMemcpy(&partPerCellCuda[initPCellId], &partPerCell[initPCellId],
                                                    sizeof(uint), hipMemcpyHostToDevice);

                auto remove_status = orderedParticlesCuda.remove_by_id(currPart.id);
                if (remove_status)
                    throw std::runtime_error("Error in remove");

                auto insert_status = orderedParticlesCuda.insert(particle, *partCellStartIdx);
                if (insert_status)
                    throw std::runtime_error("Error in insert");

                size_t cells_in_range = initPCellId > newPCellId ?
                            initPCellId - newPCellId : newPCellId - initPCellId;

                size_t threadsPerBlock = std::min(cells_in_range, MAX_BLOCK_THREADS);
                size_t numBlocks = (cells_in_range + threadsPerBlock - 1) / threadsPerBlock;

                if (initPCellId > newPCellId)
                    backward_move_kernel<<<numBlocks, threadsPerBlock>>>
                                (cellStartIdxCuda, newPCellId, cells_in_range);

                else if (initPCellId < newPCellId)
                    forward_move_kernel<<<numBlocks, threadsPerBlock>>>
                                (cellStartIdxCuda, initPCellId, cells_in_range);

                delete partCellStartIdx;
            }
            success++;
            std::cout << "Accepting" << std::endl;
        }
        else
            std::cout << "Rejecting" << std::endl;
        std::cout << std::endl;
    }
    return success;
}

void Grid::system_energy() {
    energy = 0;

    for (auto &particle: particles) {
        D3<double> p_point = particle.get_coord();
        auto pCellId = cell_id(cell(p_point));

        energy_all_cell_kernel<<<nAdjCells, maxPartPerCell2pow, maxPartPerCell2pow*sizeof(double)>>>
                        (energiesCuda, particle, partPerCellCuda, orderedParticlesCuda.get_array(),
                         cellStartIdxCuda, cudaL, cnCuda, pCellId);

        auto *energies = new double[nAdjCells];
        hipMemcpy(energies, energiesCuda, sizeof(double) * nAdjCells, hipMemcpyDeviceToHost);

        for (uint i = 0; i < nAdjCells; i++)
            energy += energies[i];

        delete[] energies;
    }
    energy /= 2.0;
}


enum paramsMLen{
    TYPE_MLEN = 6, SN_MLEN = 5, NAME_MLEN = 4, ALT_LOC_IND_MLEN = 1, RES_NAME_MLEN = 3,
    CHAIN_IND_MLEN = 1, RES_SEQ_NUM_MLEN = 4, RES_INS_CODE_MLEN = 1,
    X_MLEN = 8, Y_MLEN = 8, Z_MLEN = 8, OCC_MLEN = 6, TEMP_FACTOR_MLEN = 6,
    SEG_ID_MLEN = 4, ELEM_SYMB_MLEN = 2, CHARGE_MLEN = 2
};

static std::string
format(double fp_num, unsigned nint, unsigned nfrac) {
    auto maxNum = std::pow(10, nint);
    if (fp_num >= maxNum)
        throw std::invalid_argument(std::string("Number is too big (max ")
                                    + std::to_string(maxNum) + std::string(")"));

    fp_num = std::ceil(fp_num * maxNum) / static_cast<double>(maxNum);

    std::stringstream fp_num_ss;
    fp_num_ss.precision(nfrac);
    fp_num_ss.setf(std::ios::fixed, std::ios::floatfield);
    fp_num_ss << fp_num;

    return fp_num_ss.str();
}

constexpr auto COORD_MINT = 4;
constexpr auto COORD_MFRAC = 3;

constexpr auto OCCTEMP_MINT = 3;
constexpr auto OCCTEMP_MFRAC = 2;

static std::string
fcoord (double coord) {
    return format(coord, COORD_MINT, COORD_MFRAC);
}

static std::string
focctemp (double occtemp) {
    return format(occtemp, OCCTEMP_MINT, OCCTEMP_MFRAC);
}

enum direction{left, right};

static std::string check_fill (std::string val, size_t len, direction align) {
    auto val_len = val.size();
    if (val_len == 0)
        for (auto i = len; i > 0; i--, val += " ");
    else if (val_len > len)
        throw std::invalid_argument("Invalid argument length (too long): expected " +
                                    std::to_string(len) + ", got " + std::to_string(val_len));
    else {
        std::string xfix;
        for (auto i = val.size(); i < len; i++, xfix += " ");
        val = (align == right) ? xfix + val : val + xfix;
    }
    return val;
}

static std::string
check_fill(std::string val, int len) {
    return check_fill(std::move(val), len, left);
}

static void
export_to_pdb ( const std::string& fn,             // output filename with extension
                std::string type,           // 1-6
                std::string sn,             // 7-11  right
                std::string name,           // 13-16
                std::string alt_loc_ind,    // 17
                std::string res_name,       // 18-20 right
                std::string chain_ind,      // 22
                std::string res_seq_num,    // 23-26 right
                std::string res_ins_code,   // 27
                std::string x,              // 31-38 right
                std::string y,              // 39-46 right
                std::string z,              // 47-54 right
                std::string occ,            // 55-60 right
                std::string temp_factor,    // 61-66 right
                std::string seg_id,         // 73-76
                std::string elem_symb,      // 77-78 right
                std::string charge          // 79-80
              ){
    // Workaround
    if (stoi(sn) >= 100000)
        sn = "99999";

    type = check_fill(type, TYPE_MLEN);
    sn = check_fill(sn, SN_MLEN, right);
    name = check_fill(name, NAME_MLEN);
    alt_loc_ind = check_fill(alt_loc_ind, ALT_LOC_IND_MLEN);
    res_name = check_fill(res_name, RES_NAME_MLEN, right);
    chain_ind = check_fill(chain_ind, CHAIN_IND_MLEN);
    res_seq_num = check_fill(res_seq_num, RES_SEQ_NUM_MLEN, right);
    res_ins_code = check_fill(res_ins_code, RES_INS_CODE_MLEN);
    x = check_fill(x, X_MLEN, right);
    y = check_fill(y, Y_MLEN, right);
    z = check_fill(z, Z_MLEN, right);
    occ = check_fill(occ, OCC_MLEN, right);
    temp_factor = check_fill(temp_factor, TEMP_FACTOR_MLEN, right);
    seg_id = check_fill(seg_id, SEG_ID_MLEN);
    elem_symb = check_fill(elem_symb, ELEM_SYMB_MLEN, right);
    charge = check_fill(charge, CHARGE_MLEN);

    std::ofstream pdb_file(fn, std::ofstream::app);
    pdb_file << type << sn << " " << name << alt_loc_ind << res_name << " " << chain_ind
             << res_seq_num << res_ins_code << "   " << x << y << z << occ << temp_factor
             << "     " << elem_symb << charge << std::endl;
    pdb_file.close();
}

void Grid::export_to_pdb(const std::string& fn) {
    remove(fn.data());
    unsigned serial_num = 1;
    for (auto particle : particles) {

        std::string sn_str = std::to_string(serial_num);

        const std::string particle_type = "ATOM";
        const std::string atom_name = "C";
        const std::string sort_of_elem = std::to_string(1);
        const std::string temp_factor = focctemp(0);

        ::export_to_pdb(fn, particle_type, std::to_string(serial_num), atom_name, "", "", "",
                sort_of_elem, "", fcoord(particle.x), fcoord(particle.y), fcoord(particle.z),
                focctemp(particle.sigma), temp_factor, "", "", "");
        serial_num++;
    }
}

void Grid::export_to_cf(const std::string& fn) {
    std::ofstream cf_file(fn);
    if (!cf_file)
        throw std::runtime_error("Error while opening file for export " + fn);

    for (auto p: particles) {
        char buff[256];
        sprintf(buff, "%15ld%15ld%4d%20.10lf%20.10lf%20.10lf\n", p.id, p.id, 1, p.x, p.y, p.z);
        std::string buff_str{buff};
        cf_file << buff_str;
    }
}

void Grid::import_from_cf(const std::string& fn) {
    // import from custom format file
    std::ifstream cf_file(fn);
    if (!cf_file)
        throw std::runtime_error("Error while opening file for import " + fn);

    std::string line;

    while (std::getline(cf_file, line)) {
        std::stringstream ss(line);

        // skip first 3 columns, because they are useless for particle constructor
        long trash;
        ss >> trash >> trash >> trash;

        double x, y, z;
        ss >> x >> y >> z;
        Particle p(x, y, z, pSigma);
        complex_insert(p);
    }
    cf_file.close();

    if (particles.size() != n)
        throw std::invalid_argument("During import: too many particles in CF file.\
                Either grid is badly preconfigured or CF file is corrupted.");
}

void Grid::complex_insert(Particle p) {
    particles.push_back(p);
    auto p_cell_id = cell_id(cell(p.get_coord()));

    // Cell start index in ordered array for the current particle (which is inserted)
    uint *partCellStartIdx = new uint;
    hipMemcpy(partCellStartIdx, &cellStartIdxCuda[p_cell_id], sizeof(uint),
                                                hipMemcpyDeviceToHost);

    orderedParticlesCuda.insert(p, *partCellStartIdx);
    partPerCell[p_cell_id]++;
    hipMemcpy(&partPerCellCuda[p_cell_id], &partPerCell[p_cell_id], sizeof(uint),
                                                            hipMemcpyHostToDevice);

    if (n_cells < p_cell_id + 1)
        throw std::runtime_error("Cell_idx > number of cells, which is impossible");

    size_t N = n_cells-p_cell_id-1;
    if (N > 0) {
        size_t threadsPerBlock = std::min(N, MAX_BLOCK_THREADS);
        size_t numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;
        update_kernel<<<numBlocks, threadsPerBlock>>>(cellStartIdxCuda, p_cell_id+1, N);
    }

    delete partCellStartIdx;
}




void Grid::compute_adj_cells() {
    for (auto ix = 0; ix < dimCells.x; ix++) {
        for (auto iy = 0; iy < dimCells.y; iy++) {
            for (auto iz = 0; iz < dimCells.z; iz++) {
                auto ikx = ix;
                auto iky = iy;
                auto ikz = iz;
                auto parr_cell_id = cell_id(D3<int>(ikx, iky, ikz));
                auto k = 0;
                for (auto jx = ix-1; jx <= ix+1; jx++) {
                    for (auto jy = iy-1; jy <= iy+1; jy++) {
                        for (auto jz = iz-1; jz <= iz+1; jz++) {
                            // this cell coordinates
                            auto tc_x = jx;
                            auto tc_y = jy;
                            auto tc_z = jz;
                            if (tc_x < 0) tc_x += dimCells.x; if (tc_x > dimCells.x-1) tc_x -= dimCells.x;
                            if (tc_y < 0) tc_y += dimCells.y; if (tc_y > dimCells.y-1) tc_y -= dimCells.y;
                            if (tc_z < 0) tc_z += dimCells.z; if (tc_z > dimCells.z-1) tc_z -= dimCells.z;
                            
                            uint curr_cell_id = cell_id(D3<double>(tc_x, tc_y, tc_z));
                            cn[parr_cell_id].ac[k] = curr_cell_id;
                            k++;
                        }
                    }
                }
            }
        }
    }
}
